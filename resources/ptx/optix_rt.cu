#include "hip/hip_runtime.h"
#include <optix.h>
#include <iostream>
#include "vector.cuh"
#include <hip/hip_math_constants.h>

struct Params {
    bool    *in_mask;
    float   *in_ox, *in_oy, *in_oz,
            *in_dx, *in_dy, *in_dz,
            *in_mint, *in_maxt;
    float   *out_t, *out_u, *out_v,
            *out_ng_x, *out_ng_y, *out_ng_z,
            *out_ns_x, *out_ns_y, *out_ns_z,
            *out_p_x, *out_p_y, *out_p_z,
            *out_dp_du_x, *out_dp_du_y, *out_dp_du_z,
            *out_dp_dv_x, *out_dp_dv_y, *out_dp_dv_z;

    unsigned long long *out_shape_ptr;
    unsigned int *out_primitive_id;

    bool *out_hit;

    OptixTraversableHandle handle;
    bool rg_any;
};


extern "C" {
__constant__ Params params;
}

struct HitGroupData {
    unsigned long long shape_ptr;
    Vector3ui* faces;
    Vector3f* vertex_positions;
    Vector3f* vertex_normals;
    Vector2f* vertex_texcoords;
};

__forceinline__ __device__ float3 make_float3(const Vector3f& v) {
    return make_float3(v.x(), v.y(), v.z());
}
__forceinline__ __device__ Vector3f make_Vector3f(const float3& v) {
    return Vector3f(v.x, v.y, v.z);
}

__device__ void coordinate_system(Vector3f n, Vector3f &x, Vector3f &y) {
    /* Based on "Building an Orthonormal Basis, Revisited" by
       Tom Duff, James Burgess, Per Christensen,
       Christophe Hery, Andrew Kensler, Max Liani,
       and Ryusuke Villemin (JCGT Vol 6, No 1, 2017) */

    float s = copysignf(1.f, n.z()),
          a = -1.f / (s + n.z()),
          b = n.x() * n.y() * a;

    x = Vector3f(n.x() * n.x() * a * s + 1.f, b * s, -n.x() * s);
    y = Vector3f(b, s + n.y() * n.y() * a, -n.y());
}

__device__ void ray_attr(
    const HitGroupData* sbt_data,
    Vector3f &p,
    Vector2f &uv,
    Vector3f &ns,
    Vector3f &ng,
    Vector3f &dp_du,
    Vector3f &dp_dv) {
    float2 float2_uv = optixGetTriangleBarycentrics();
    uv = Vector2f(float2_uv.x, float2_uv.y);
    float uv0 = 1.f - uv.x() - uv.y(),
          uv1 = uv.x(),
          uv2 = uv.y();

    const Vector3ui* faces            = sbt_data->faces;
    const Vector3f* vertex_positions  = sbt_data->vertex_positions;
    const Vector3f* vertex_normals    = sbt_data->vertex_normals;
    const Vector2f* vertex_texcoords  = sbt_data->vertex_texcoords;

    Vector3ui face = faces[optixGetPrimitiveIndex()];

    Vector3f p0 = vertex_positions[face.x()],
             p1 = vertex_positions[face.y()],
             p2 = vertex_positions[face.z()];

    Vector3f dp0 = p1 - p0,
             dp1 = p2 - p0;

    p = p0 * uv0 + p1 * uv1 + p2 * uv2;

    ng = normalize(cross(dp0, dp1));
    coordinate_system(ng, dp_du, dp_dv);

    if (vertex_normals != nullptr) {
        Vector3f n0 = vertex_normals[face.x()],
                 n1 = vertex_normals[face.y()],
                 n2 = vertex_normals[face.z()];

        ns = normalize(n0 * uv0 + n1 * uv1 + n2 * uv2);
    } else {
        ns = ng;
    }

    if (vertex_texcoords != nullptr) {
        Vector2f t0 = vertex_texcoords[face.x()],
                 t1 = vertex_texcoords[face.y()],
                 t2 = vertex_texcoords[face.z()];

        uv = t0 * uv0 + t1 * uv1 + t2 * uv2;

        Vector2f dt0 = t1 - t0, dt1 = t2 - t0;
        float det = dt0.x() * dt1.y() - dt0.y() * dt1.x();

        if (det != 0.f) {
            float inv_det = 1.f / det;
            dp_du = ( dt1.y() * dp0 - dt0.y() * dp1) * inv_det;
            dp_dv = (-dt1.x() * dp0 + dt0.x() * dp1) * inv_det;
        }
    }
}

extern "C" __global__ void __raygen__rg() {
    uint3 launch_dims = optixGetLaunchDimensions();
    uint3 launch_index3 = optixGetLaunchIndex();
    unsigned int launch_index = launch_index3.x + (launch_index3.y + launch_index3.z * launch_dims.y) * launch_dims.x;

    Vector3f ro = Vector3f(params.in_ox[launch_index],
                           params.in_oy[launch_index],
                           params.in_oz[launch_index]),
             rd = Vector3f(params.in_dx[launch_index],
                           params.in_dy[launch_index],
                           params.in_dz[launch_index]);
    float  mint = params.in_mint[launch_index],
           maxt = params.in_maxt[launch_index];

    if (params.rg_any) {
        if (!params.in_mask[launch_index]) {
            params.out_hit[launch_index] = false; // TODO: check if out_hit valid??
        } else {
            optixTrace(
                params.handle,
                make_float3(ro), make_float3(rd),
                mint, maxt, 0.0f,
                OptixVisibilityMask( 1 ),
                OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
                0, 1, 0
                );
        }
    } else {
        if (!params.in_mask[launch_index]) {
            params.out_shape_ptr[launch_index] = 0;
            params.out_t[launch_index] = HIP_INF_F;
        } else {
            optixTrace(
                params.handle,
                make_float3(ro), make_float3(rd),
                mint, maxt, 0.0f,
                OptixVisibilityMask( 1 ),
                OPTIX_RAY_FLAG_NONE,
                0, 1, 0
                );
        }
    }
}

__device__ inline float squared_norm(Vector3f v) {
    return dot(v, v);
}

extern "C" __global__ void __closesthit__ch() {
    uint3 launch_dims = optixGetLaunchDimensions();
    uint3 launch_index3 = optixGetLaunchIndex();
    unsigned int launch_index = launch_index3.x + (launch_index3.y + launch_index3.z * launch_dims.y) * launch_dims.x;

    if (params.out_hit != nullptr) {
        params.out_hit[launch_index] = true;
    } else {
        Vector3f p;
        Vector2f uv;
        Vector3f ns;
        Vector3f ng;
        Vector3f dp_du;
        Vector3f dp_dv;
        const HitGroupData* sbt_data = (HitGroupData*)optixGetSbtDataPointer();

        ray_attr(sbt_data, p, uv, ns, ng, dp_du, dp_dv);

        params.out_shape_ptr[launch_index] = sbt_data->shape_ptr;

        params.out_primitive_id[launch_index] = optixGetPrimitiveIndex();

        params.out_u[launch_index] = uv.x();
        params.out_v[launch_index] = uv.y();

        params.out_ng_x[launch_index] = ng.x();
        params.out_ng_y[launch_index] = ng.y();
        params.out_ng_z[launch_index] = ng.z();

        params.out_ns_x[launch_index] = ns.x();
        params.out_ns_y[launch_index] = ns.y();
        params.out_ns_z[launch_index] = ns.z();

        params.out_p_x[launch_index] = p.x();
        params.out_p_y[launch_index] = p.y();
        params.out_p_z[launch_index] = p.z();

        params.out_dp_du_x[launch_index] = dp_du.x();
        params.out_dp_du_y[launch_index] = dp_du.y();
        params.out_dp_du_z[launch_index] = dp_du.z();

        params.out_dp_dv_x[launch_index] = dp_dv.x();
        params.out_dp_dv_y[launch_index] = dp_dv.y();
        params.out_dp_dv_z[launch_index] = dp_dv.z();

        Vector3f ray_o = make_Vector3f(optixGetWorldRayOrigin());
        Vector3f ray_d = make_Vector3f(optixGetWorldRayDirection());

        params.out_t[launch_index] = sqrt(squared_norm(p - ray_o) / squared_norm(ray_d));
    }
}

extern "C" __global__ void __miss__ms() {
    uint3 launch_dims = optixGetLaunchDimensions();
    uint3 launch_index3 = optixGetLaunchIndex();
    unsigned int launch_index = launch_index3.x + (launch_index3.y + launch_index3.z * launch_dims.y) * launch_dims.x;

    if (params.out_hit != nullptr) {
        params.out_hit[launch_index] = false;
    } else {
        params.out_shape_ptr[launch_index] = 0;
        params.out_t[launch_index] = HIP_INF_F;
    }
}

struct OptixException
{
    int code;
    const char* string;
};

__constant__ OptixException exceptions[] = {
    { OPTIX_EXCEPTION_CODE_STACK_OVERFLOW, "OPTIX_EXCEPTION_CODE_STACK_OVERFLOW" },
    { OPTIX_EXCEPTION_CODE_TRACE_DEPTH_EXCEEDED, "OPTIX_EXCEPTION_CODE_TRACE_DEPTH_EXCEEDED" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_DEPTH_EXCEEDED, "OPTIX_EXCEPTION_CODE_TRAVERSAL_DEPTH_EXCEEDED" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_TRAVERSABLE, "OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_TRAVERSABLE" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_MISS_SBT, "OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_MISS_SBT" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_HIT_SBT, "OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_HIT_SBT" }
};

extern "C" __global__ void __exception__err() {
    int ex_code = optixGetExceptionCode();
    printf("Optix Exception %u: %s\n", ex_code, exceptions[ex_code].string);
    // TODO: retreive more informations based on exception
}
