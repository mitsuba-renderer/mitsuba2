#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>

#include <mitsuba/render/optix/matrix.cuh>
#include <mitsuba/render/optix/common.h>

// Include all shapes CUDA headers to generate their PTX programs
#include <mitsuba/render/optix/shapes.h>

extern "C" __global__ void __raygen__rg() {
    unsigned int launch_index = calculate_launch_index();

    Vector3f ro = Vector3f(params.in_ox[launch_index],
                           params.in_oy[launch_index],
                           params.in_oz[launch_index]),
             rd = Vector3f(params.in_dx[launch_index],
                           params.in_dy[launch_index],
                           params.in_dz[launch_index]);
    float mint = params.in_mint[launch_index],
          maxt = params.in_maxt[launch_index];

    if (params.out_hit != nullptr) {
        if (!params.in_mask[launch_index]) {
            params.out_hit[launch_index] = false;
        } else {
            optixTrace(
                params.handle,
                make_float3(ro), make_float3(rd),
                mint, maxt, 0.0f,
                OptixVisibilityMask( 1 ),
                OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
                0, 1, 0
                );
        }
    } else {
        if (!params.in_mask[launch_index]) {
            params.out_shape_ptr[launch_index] = 0;
            params.out_t[launch_index] = HIP_INF_F;
        } else {
            optixTrace(
                params.handle,
                make_float3(ro), make_float3(rd),
                mint, maxt, 0.0f,
                OptixVisibilityMask( 1 ),
                OPTIX_RAY_FLAG_NONE,
                0, 1, 0
                );
        }
    }
}

extern "C" __global__ void __miss__ms() {
    unsigned int launch_index = calculate_launch_index();

    if (params.out_hit != nullptr) {
        params.out_hit[launch_index] = false;
    } else {
        params.out_shape_ptr[launch_index] = 0;
        params.out_t[launch_index] = HIP_INF_F;
    }
}

struct OptixException {
    int code;
    const char* string;
};

__constant__ OptixException exceptions[] = {
    { OPTIX_EXCEPTION_CODE_STACK_OVERFLOW, "OPTIX_EXCEPTION_CODE_STACK_OVERFLOW" },
    { OPTIX_EXCEPTION_CODE_TRACE_DEPTH_EXCEEDED, "OPTIX_EXCEPTION_CODE_TRACE_DEPTH_EXCEEDED" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_DEPTH_EXCEEDED, "OPTIX_EXCEPTION_CODE_TRAVERSAL_DEPTH_EXCEEDED" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_TRAVERSABLE, "OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_TRAVERSABLE" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_MISS_SBT, "OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_MISS_SBT" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_HIT_SBT, "OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_HIT_SBT" }
};

extern "C" __global__ void __exception__err() {
    int ex_code = optixGetExceptionCode();
    printf("Optix Exception %u: %s\n", ex_code, exceptions[ex_code].string);
    // TODO: retreive more informations based on exception
}
